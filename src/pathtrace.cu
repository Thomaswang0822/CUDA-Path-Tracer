#include "hip/hip_runtime.h"
#include "pathtrace.h"
#include "materials.h"
#include "sampler.h"
#include "scene.h"
#include "sceneStructs.h"
#include "utilities.h"
#include <hip/hip_runtime.h>
#include <>  // "let VS know" blockIdx etc.
#include <thrust/device_ptr.h>
#include <thrust/remove.h>


#define PixelIdxForTerminated -1
static Scene* hst_scene = nullptr;
static GuiDataContainer* guiData = nullptr;
static glm::vec3* dev_image = nullptr;
static Intersection* dev_intersections = nullptr;
// One for running kernels, the other for storage
static PathSegment* paths_alive = nullptr;
static PathSegment* paths_done = nullptr;
static thrust::device_ptr<PathSegment> thr_paths_alive;
static thrust::device_ptr<PathSegment> thr_paths_done;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&paths_alive, pixelcount * sizeof(PathSegment));
	thr_paths_alive = thrust::device_ptr<PathSegment>(paths_alive);
	hipMalloc(&paths_done, pixelcount * sizeof(PathSegment));
	thr_paths_done = thrust::device_ptr<PathSegment>(paths_done);

	hipMalloc(&dev_intersections, pixelcount * sizeof(Intersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(Intersection));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(paths_alive);
	hipFree(paths_done);
	hipFree(dev_intersections);
	
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(const Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		//segment.ray.origin = cam.position;
		segment.throughput = glm::vec3(1.0f);
		segment.radiance = glm::vec3(0.f);

		// antialiasing by jittering the ray
		/*segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);*/
		Sampler sampler(iter, index, 0);
		cam.generateCameraRay(segment.ray, x, y, sampler.sample2D());

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth,
	int num_paths,
	PathSegment* pathSegments,
	DevScene* scene,
	Intersection* intersections
) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths) {
		PathSegment& pathSegment = pathSegments[path_index];
#if BVH_DEBUG_VISUALIZATION
		scene->visualizedIntersect(pathSegment.ray, intersections[path_index]);
#else
		Intersection intersec;
		scene->intersect(pathSegment.ray, intersec);

		if (intersec.primId != NullPrimitive) {
			if (scene->devMaterials[intersec.materialId].type == Material::Type::Light) {
#if SCENE_LIGHT_SINGLE_SIDED
				if (glm::dot(intersec.normal, pathSegment.ray.direction) < 0.f) {
					intersec.primId = NullPrimitive;
				}
				else
#endif // SCENE_LIGHT_SINGLE_SIDED
					if (depth != 0) {
						// If not first ray, preserve previous sampling information for
						// MIS calculation
						intersec.prevPos = pathSegment.ray.origin;
						// intersec.prevBSDFPdf = segment.BSDFPdf;
					}
			}
			else {
				intersec.wo = -pathSegment.ray.direction;
			}
		}
		intersections[path_index] = intersec;
#endif // BVH_DEBUG_VISUALIZATION
	}  // end if (path_index < num_paths)
}

__global__ void shadeSegment(
	int iter,
	int depth,
	int numPaths,
	Intersection* intersections,
	PathSegment* segments,
	DevScene* scene
) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	// Should be unnecessary with stream compaction implemented
	if (idx >= numPaths) {
		return;
	}

	// Deal with miss
	Intersection& intersec = intersections[idx];
	if (intersec.primId == NullPrimitive) {
		segments[idx].remainingBounces = 0;
		return;
	}

#if BVH_DEBUG_VISUALIZATION
	float logDepth = 0.f;
	int size = scene->BVHSize;
	while (size) {
		logDepth += 1.f;
		size >>= 1;
	}
	segment.radiance = glm::vec3(float(intersec.primId) / logDepth * .1f);
	//segment.radiance = intersec.primId > 16 ? glm::vec3(1.f) : glm::vec3(0.f);
	segment.remainingBounces = 0;
	return;
#endif

	Sampler sampler(iter, idx, segments[idx].remainingBounces);
	const Material& material = scene->devMaterials[intersec.materialId];
	PathSegment& segment = segments[idx];
	glm::vec3 accRadiance(0.f);

	//bool deltaBSDF = material.type == Material::Type::Dielectric;

	/// If hit a light source
	if (material.type == Material::Type::Light) {
		glm::vec3 radiance = material.baseColor * material.emittance;
		if (depth == 0 || segment.isDeltaSample) {
			// If this is the first bounce or if we just had a specular bounce
			accRadiance += radiance * segment.throughput;
		}
		else {
			/// previous shading point (ray.origin) bounced off to a light
			/// so we do MIS
			float lightPdf = Math::pdfAreaToSolidAngle(Math::luminance(radiance) * scene->sumLightPowerInv,
				intersec.prevPos, intersec.position, intersec.normal);
			float BSDFPdf = segment.BSDFpdf;
			accRadiance += radiance * segment.throughput * Math::powerHeuristic(BSDFPdf, lightPdf);
		}
		// stop bouncing in both cases
		segment.remainingBounces = 0;
	}
	/// Do MIS
	else {
		bool deltaBSDF = (material.type == Material::Type::Dielectric);
		if (!deltaBSDF && glm::dot(intersec.normal, intersec.wo) < 0.f) {
			// other than Dielectric (glass), we don't allow refraction.
			intersec.normal = -intersec.normal;
		}

		// Light Sampling: accumulate radiance right away
		if (!deltaBSDF) {
			glm::vec3 radiance;
			glm::vec3 wi;
			float lightPdf = scene->sampleDirectLight(intersec.position, sampler.sample4D(), radiance, wi);

			if (lightPdf > 0.f) {
				float BSDFPdf = material.pdf(intersec.normal, intersec.wo, wi);
				accRadiance += segment.throughput *
					material.BSDF(intersec.normal, intersec.wo, wi) *
					radiance *
					Math::nonnegativeDot(intersec.normal, wi) /
					lightPdf * Math::powerHeuristic(lightPdf, BSDFPdf);
			}
		}

		// BSDF sampling: postpone to next bounce
		BSDFSample sample;
		material.sample(intersec.normal, intersec.wo, sampler.sample3D(), sample);

		if (sample.type == BSDFSampleType::Invalid) {
			// Terminate path if sampling fails
			segment.remainingBounces = 0;
		}
		else {
			bool deltaSample = (sample.type & BSDFSampleType::Specular);
			segment.throughput *= sample.bsdf / sample.pdf *
				(deltaSample ? 1.f : Math::absDot(intersec.normal, sample.dir));
			segment.ray = Ray::makeOffsetRay(intersec.position, sample.dir);
			segment.BSDFpdf = sample.pdf;
			segment.isDeltaSample = deltaSample;
			segment.remainingBounces--;
		}
	}
	segment.radiance += accRadiance;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		const PathSegment& iterationPath = iterationPaths[index];
		//image[iterationPath.pixelIndex] += iterationPath.throughput;
		if (iterationPath.pixelIndex >= 0 && iterationPath.remainingBounces <= 0) {
			if (Debug::isNanInf(iterationPath.radiance)) {
				return;
			}
			image[iterationPath.pixelIndex] += glm::clamp(iterationPath.radiance,
				glm::vec3(0.f), glm::vec3(1e6f));
		}
	}
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 colorRawAvg = image[index] / float(iter);

		// Do ACES tone mapping and Gamma correction
		glm::vec3 colorOut = Math::correctGamma(Math::mapACES(colorRawAvg));
		glm::vec3 intColor = glm::clamp(glm::ivec3(colorOut * 255.f), glm::ivec3(0.f), glm::ivec3(255.f));

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = intColor.x;
		pbo[index].y = intColor.y;
		pbo[index].z = intColor.z;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {

	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y
	);

	// 1D block for path tracing
	const int blockSize1d = 128;

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d>>> (cam, iter, traceDepth, paths_alive);
	checkCUDAError("generateRayFromCamera");
	hipDeviceSynchronize();

	// increment each iteration
	int depth = 0;
	int num_paths = pixelcount;

	/// @note With stream compaction, thr_paths_done points to the start of terminated paths data.
	/// And we also need a running pointer that gives the next clean memory address,
	/// which enables more data being written in the next iteration.
	auto next_thr_paths_done = thr_paths_done;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(Intersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
			depth,
			num_paths,
			paths_alive,  // PathSegment*
			hst_scene->devScene,
			dev_intersections
		);
		checkCUDAError("computeIntersections");
		hipDeviceSynchronize();
		//depth++;

		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// TODO:
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.

		//shadeFakeMaterial <<<numblocksPathSegmentTracing, blockSize1d>>> (iter, num_paths, dev_intersections, paths_alive, dev_materials);
		shadeSegment << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, 
			depth,
			num_paths, 
			dev_intersections, 
			paths_alive, 
			hst_scene->devScene
		);
		checkCUDAError("shadeSegment");
		hipDeviceSynchronize();

		/**
		 * @brief Compact paths that are terminated but carry contribution into a separate buffer.
		 * It copies to next_thr_paths_done and advance it to next clean memory address, but thr_paths_alive isn't shortened.
		 * 
		 * @see https://nvidia.github.io/cccl/thrust/api/function_group__stream__compaction_1gaeec02acfde68e411ca7d09063241f4d7.html#thrust-remove-copy-if.
		 */
		next_thr_paths_done = thrust::remove_copy_if(thr_paths_alive, thr_paths_alive + num_paths, next_thr_paths_done, CompactTerminatedPaths());
		// Remove paths that yield no contribution
		/**
		 * @brief Remove paths that yield no contribution.
		 * 
		 * @see https://nvidia.github.io/cccl/thrust/api/function_group__stream__compaction_1gaf01d45b30fecba794afae065d625f94f.html#thrust-remove-if
		 */
		auto thr_paths_alive_end = thrust::remove_if(thr_paths_alive, thr_paths_alive + num_paths, RemoveInvalidPaths());
		num_paths = static_cast<int>(thr_paths_alive_end - thr_paths_alive);
		//std::cout << "Remaining paths: " << num_paths << "\n";

		iterationComplete = bool(num_paths == 0);
		depth++;

		if (guiData != nullptr)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	int numEffectivePaths = static_cast<int>(next_thr_paths_done.get() - paths_done);
	finalGather <<<numBlocksPixels, blockSize1d >>> (numEffectivePaths, dev_image, paths_done);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO <<<blocksPerGrid2d, blockSize2d>>> (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	Cuda::memcpyDevToHost(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3));

	checkCUDAError("pathtrace");
}
