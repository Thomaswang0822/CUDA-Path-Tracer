#include "hip/hip_runtime.h"
#include "pathtrace.h"
#include "materials.h"
#include "sampler.h"
#include "scene.h"
#include "sceneStructs.h"
#include "utilities.h"
#include <hip/hip_runtime.h>
#include <>  // "let VS know" blockIdx etc.
#include <thrust/device_ptr.h>
#include <thrust/remove.h>


//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 colorRawAvg = image[index] / float(iter);

		// Do ACES tone mapping and Gamma correction
		glm::vec3 colorOut = mathUtil::correctGamma(mathUtil::mapACES(colorRawAvg));
		glm::vec3 intColor = glm::clamp(glm::ivec3(colorOut * 255.f), glm::ivec3(0.f), glm::ivec3(255.f));

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = intColor.x;
		pbo[index].y = intColor.y;
		pbo[index].z = intColor.z;
	}
}

#define PixelIdxForTerminated -1
static Scene* hst_scene = nullptr;
static GuiDataContainer* guiData = nullptr;
static glm::vec3* dev_image = nullptr;
static Intersection* dev_intersections = nullptr;
// One for running kernels, the other for storage
static PathSegment* paths_alive = nullptr;
static PathSegment* paths_done = nullptr;
static thrust::device_ptr<PathSegment> thr_paths_alive;
static thrust::device_ptr<PathSegment> thr_paths_done;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&paths_alive, pixelcount * sizeof(PathSegment));
	thr_paths_alive = thrust::device_ptr<PathSegment>(paths_alive);
	hipMalloc(&paths_done, pixelcount * sizeof(PathSegment));
	thr_paths_done = thrust::device_ptr<PathSegment>(paths_done);

	hipMalloc(&dev_intersections, pixelcount * sizeof(Intersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(Intersection));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(paths_alive);
	hipFree(paths_done);
	hipFree(dev_intersections);
	
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		//segment.ray.origin = cam.position;
		segment.throughput = glm::vec3(1.0f);
		segment.radiance = glm::vec3(0.f);

		// antialiasing by jittering the ray
		/*segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);*/
		Sampler sampler(iter, index, 0);
		cam.generateCameraRay(segment.ray, x, y, sampler.sample2D());

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, DevScene* scene
	, Intersection* intersections
)
{
	// Turned to BVH traversal

	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths) {
		PathSegment pathSegment = pathSegments[path_index];
#if BVH_DEBUG_VISUALIZATION
		scene->visualizedIntersect(pathSegment.ray, intersections[path_index]);
#else
		scene->intersect(pathSegment.ray, intersections[path_index]);
#endif // BVH_DEBUG_VISUALIZATION

		
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a Intersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, Intersection* intersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		Intersection intersection = intersections[idx];
		if (intersection.primitive == NullPrimitive) { // if the intersection exists...
			// Set up the Sampler
			Sampler sampler(iter, idx, 0);

			Material& material = materials[intersection.materialId];
			glm::vec3 materialColor = material.baseColor;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].throughput *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.normal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].throughput *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.materialId * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].throughput *= sampler.sample1D(); // apply some noise because why not
			}
		}
		else {
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
			pathSegments[idx].throughput = glm::vec3(0.0f);
		}
	}
}

__global__ void shadeSegment(
	int iter,
	int numPaths,
	Intersection* intersections,
	PathSegment* segments,
	DevScene* scene
) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= numPaths) {
		return;
	}
	Intersection intersec = intersections[idx];
	if (intersec.primitive == NullPrimitive) {
		//segments[idx].throughput = glm::vec3(0.0f);
		segments[idx].pixelIndex = PixelIdxForTerminated;
		return;
	}

	Sampler sampler(iter, idx, segments[idx].remainingBounces);
	Material& material = scene->devMaterials[intersec.materialId];
	PathSegment& segment = segments[idx];

	// TODO
	// Perform light area sampling and MIS
	//segment.radiance = material.baseColor;

#if BVH_DEBUG_VISUALIZATION
	float logDepth = 0.f;
	int size = scene->BVHSize;
	while (size) {
		logDepth += 1.f;
		size >>= 1;
	}
	segment.radiance = glm::vec3(float(intersec.primitive) / logDepth * .1f);
	//segment.radiance = intersec.primitive > 16 ? glm::vec3(1.f) : glm::vec3(0.f);
	segment.remainingBounces = 0;
	return;
#endif

	if (material.type == Material::Type::Light) {
		// TODO
		// MIS

		//segment.throughput *= material.baseColor * material.emittance;
		segment.radiance += segment.throughput * material.baseColor * material.emittance;
		segment.remainingBounces = 0;
	}
	else {
		/*/// DEBUG
		segment.radiance = intersec.normal;
		segment.remainingBounces = 0;
		return;*/

		if (material.type != Material::Type::Dielectric && glm::dot(intersec.normal, intersec.inDir) < 0.f) {
			intersec.normal = -intersec.normal;
		}

		BSDFSample sample;
		materialSample(intersec.normal, intersec.inDir, material, sampler.sample3D(), sample);

		if (sample.type == BSDFSampleType::Invalid) {
			// Terminate path if sampling fails
			//segment.radiance = DEBUG_RED;
			segment.remainingBounces = 0;
		}
		/// DEBUG
		/*else if (iter > 0) {
			segment.radiance = sample.bsdf / sample.pdf;
			segment.remainingBounces = 0;
			return;
		}*/
		else {
			bool isSampleDelta = (sample.type & BSDFSampleType::Specular);
			segment.throughput *= sample.bsdf / sample.pdf *
				(isSampleDelta ? 1.f : mathUtil::absDot(intersec.normal, sample.dir));
			segment.ray.nextRay(intersec.position, sample.dir);
			segment.remainingBounces--;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		//image[iterationPath.pixelIndex] += iterationPath.throughput;
		if (iterationPath.pixelIndex >= 0 && iterationPath.remainingBounces == 0) {
			image[iterationPath.pixelIndex] += iterationPath.radiance;
		}
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {

	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y
	);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d>>> (cam, iter, traceDepth, paths_alive);
	checkCUDAError("generateRayFromCamera");
	hipDeviceSynchronize();

	// increment each iteration
	int depth = 0;
	int num_paths = pixelcount;

	/// @note With stream compaction, thr_paths_done points to the start of terminated paths data.
	/// And we also need a running pointer that gives the next clean memory address,
	/// which enables more data being written in the next iteration.
	auto next_thr_paths_done = thr_paths_done;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(Intersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
			depth,
			num_paths,
			paths_alive,  // PathSegment*
			hst_scene->devScene,
			dev_intersections
		);
		checkCUDAError("computeIntersections");
		hipDeviceSynchronize();
		depth++;

		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// TODO:
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.

		//shadeFakeMaterial <<<numblocksPathSegmentTracing, blockSize1d>>> (iter, num_paths, dev_intersections, paths_alive, dev_materials);
		shadeSegment << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter, 
			num_paths, 
			dev_intersections, 
			paths_alive, 
			hst_scene->devScene
		);
		checkCUDAError("shadeSegment");
		hipDeviceSynchronize();

		/**
		 * @brief Compact paths that are terminated but carry contribution into a separate buffer.
		 * It copies to next_thr_paths_done and advance it to next clean memory address, but thr_paths_alive isn't shortened.
		 * 
		 * @see https://nvidia.github.io/cccl/thrust/api/function_group__stream__compaction_1gaeec02acfde68e411ca7d09063241f4d7.html#thrust-remove-copy-if.
		 */
		next_thr_paths_done = thrust::remove_copy_if(thr_paths_alive, thr_paths_alive + num_paths, next_thr_paths_done, CompactTerminatedPaths());
		// Remove paths that yield no contribution
		/**
		 * @brief Remove paths that yield no contribution.
		 * 
		 * @see https://nvidia.github.io/cccl/thrust/api/function_group__stream__compaction_1gaf01d45b30fecba794afae065d625f94f.html#thrust-remove-if
		 */
		auto thr_paths_alive_end = thrust::remove_if(thr_paths_alive, thr_paths_alive + num_paths, RemoveInvalidPaths());
		num_paths = static_cast<int>(thr_paths_alive_end - thr_paths_alive);
		//std::cout << "Remaining paths: " << num_paths << "\n";

		iterationComplete = bool(num_paths == 0);

		if (guiData != nullptr)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	int numEffectivePaths = static_cast<int>(next_thr_paths_done.get() - paths_done);
	finalGather <<<numBlocksPixels, blockSize1d >>> (numEffectivePaths, dev_image, paths_done);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO <<<blocksPerGrid2d, blockSize2d>>> (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
